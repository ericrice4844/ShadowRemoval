#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////
//  
//      Color_Converter.cu
//      Constaints kernel functions for each of the color converters
//      
//
///////////////////////////////////////////////////////////////

#include "Color_Convert_Kernel.h"


#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime_api.h>

// ##################################################################################################
// ###   convertRGBToGrayAndHSV()   ### 
// Global implementation of the RGB to Grayscale and RGB to HSV color conversions
__global__ void 
convertRGBToGrayAndHSV_global(unsigned char* rgbImage, 
                       unsigned char* grayImage,
                       unsigned char* hsvImage,
                       int width, 
                       int height, 
                       int channels)
{
	// Row X and Column Y pixel positions
	int Col = threadIdx.x + blockIdx.x * blockDim.x;
	int Row = threadIdx.y + blockIdx.y * blockDim.y;

	// Only convert pixels in the image bounds
	if (Col < width && Row < height)
	{
		// Pixel Addresses
		int grayAddr = Row * width + Col;
		int colorAddr = channels * grayAddr;

		float R = rgbImage[colorAddr];
		float G = rgbImage[colorAddr + 1];
		float B = rgbImage[colorAddr + 2];

		// Convert color to grayscale with selected conversion factors
		grayImage[grayAddr] = R * 0.21f +   // Red
						      G * 0.71f +   // Green
						      B * 0.07f;    // Blue 

		// Initialize Max and Min values used for the HSV calculations
		float Cmax = 0;
		float Cmin = 0;

		// Normalize RGB values between 0 - 1
		R = R/255.0;
		G = G/255.0;
		B = B/255.0;

		// Determine the largest and smallest value between RGB
		if ((R > G) && (R > B) ){
			Cmax = R;
			if (G >= B){
				Cmin = B;
			}
			else{
				Cmin = G;
			}
		}
		else if ((G > R) && (G > B) ){
			Cmax = G;
			if (R >= B){
				Cmin = B;
			}
			else{
				Cmin = R;
			}
		}
		else if ((B > R) && (B > G) ){
			Cmax = B;
			if (R >= G){
				Cmin = G;
			}
			else{
				Cmin = R;
			}
		}
		else if ((R == G) && (R > B) ){
			Cmax = R;
			Cmin = B;
		}
		else if ((R == B) && (R > G) ){
			Cmax = R;
			Cmin = G;
		}
		else if ((G == B) && (G > R) ){
			Cmax = G;
			Cmin = R;
		}
		else{
			Cmax = R;
			Cmin = R;
		}
		
		// Initialize delta and H,S, and V
		float delta = Cmax - Cmin;
		float H = 0;
		float S = 0;
		float V = Cmax;

		// Compute the Hue
		if (delta == 0){
			H = 0;
		}
		else if (R == Cmax){
			H = ((60.0*((G-B)/delta)));
		}
		else if (G == Cmax){
			H = (60.0*((B-R)/delta)+120);
		}
		else {
			H = (60.0*((R-G)/delta)+240);
		}

		// Compute the Saturation
		if (V == 0){
			S = 0;
		}
		else {
			S = (delta/Cmax);
		}

		// Wrtie the HSV results back to memory
		hsvImage[colorAddr + 0] = H/2;
		hsvImage[colorAddr + 1] = S*255;
		hsvImage[colorAddr + 2] = V*255;
	}
}

// ##################################################################################################
// ###   convertRGBToGrayAndHSV()    ###
// This function sets up the device memory, calls the kernel, and retrieves the output from the device
// currently hardcoded to a specific image size 
void convertRGBToGrayAndHSV(unsigned char* hostRgbImage, 
                                       unsigned char* hostGrayImage, 
                                       unsigned char* hostHsvImage,
                                       int imageWidth, 
                                       int imageHeight, 
                                       int channels)
{

	// Initialize timing variables
	hipProfilerStart();
	hipEvent_t start0, start1, start1B, start2, start3, start4, stop0, stop1, stop1B, stop2, stop3, stop4;
	float T0, T1, T1B, T2, T3, T4;

	hipEventCreate(&start0);
	hipEventCreate(&stop0);
	hipEventRecord(start0);

	unsigned char* deviceRgbImage;
	unsigned char* deviceGrayImage;
    unsigned char* deviceHsvImage;

	// Start recording the time for device memory allocation 
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	hipEventRecord(start1);

    // Allocate device RGB image memory
	if (hipMalloc((void**)&deviceRgbImage, imageWidth * imageHeight * channels * sizeof(unsigned char)) != hipSuccess)
	{
		std::cout << "        Error!";
		return;
	}

	// Allocate device Gray image memory
	if (hipMalloc((void**)&deviceGrayImage, imageWidth * imageHeight * sizeof(unsigned char)) != hipSuccess)
	{
		hipFree(deviceRgbImage);
		std::cout << "        Error!";
		return;
	}

	// Allocate device HSV image memory
	if (hipMalloc((void**)&deviceHsvImage, imageWidth * imageHeight * channels * sizeof(unsigned char)) != hipSuccess)
	{
		hipFree(deviceRgbImage);
		std::cout << "        Error!";
		return;
	}

	// Stop recording the time for device memory allocation 
	hipEventRecord(stop1);
	hipEventSynchronize(stop1);
	hipEventElapsedTime(&T1, start1, stop1);

	// Start recording the time it takes to copy the RGB image from the host to the device
	hipEventCreate(&start1B);
	hipEventCreate(&stop1B);
	hipEventRecord(start1B);

	// copy RGB image to the device global memory
	if (hipMemcpy(deviceRgbImage, hostRgbImage, imageWidth * imageHeight * channels * sizeof(unsigned char), hipMemcpyHostToDevice) != hipSuccess)
	{
		hipFree(deviceRgbImage);
		hipFree(deviceGrayImage);
		hipFree(deviceHsvImage);
		std::cout << "        Error!";
		return;
	}

	// Stop recording the time it takes to copy the RGB image from the host to the device
	hipEventRecord(stop1B);
	hipEventSynchronize(stop1B);
	hipEventElapsedTime(&T1B, start1B, stop1B);

	// Define the grid and block size
	int blockSize = 32;
	dim3 DimGrid((imageWidth - 1) / blockSize + 1, (imageHeight - 1) / blockSize + 1, 1);
	dim3 DimBlock(blockSize, blockSize, 1);

	// Start recording the color conversion main kernel
	hipEventCreate(&start2);
	hipEventCreate(&stop2);
	hipEventRecord(start2);

	// Call the kernel that does the Grayscale and HSV conversion
	convertRGBToGrayAndHSV_global <<<DimGrid,DimBlock>>>(deviceRgbImage, 
                                                  deviceGrayImage, 
                                                  deviceHsvImage, 
                                                  imageWidth, 
                                                  imageHeight, 
                                                  channels);

	// Stop recording the color conversion main kernel
	hipEventRecord(stop2);
	hipEventSynchronize(stop2);
	hipEventElapsedTime(&T2, start2, stop2);
    
	// Start recording the time it copy the images from global memory back to the host
	hipEventCreate(&start3);
	hipEventCreate(&stop3);
	hipEventRecord(start3);	

	if (hipMemcpy(hostGrayImage, deviceGrayImage, imageWidth * imageHeight * sizeof(unsigned char), hipMemcpyDeviceToHost) != hipSuccess)
	{
		std::cout << "        Error!";
		hipFree(deviceRgbImage);
		hipFree(deviceGrayImage);
        hipFree(deviceHsvImage);
		return;
	}

	if (hipMemcpy(hostHsvImage, deviceHsvImage, imageWidth * imageHeight * channels * sizeof(unsigned char), hipMemcpyDeviceToHost) != hipSuccess)
	{
		std::cout << "        Error!";
		hipFree(deviceRgbImage);
        hipFree(deviceGrayImage);
		hipFree(deviceHsvImage);
		return;
	}

	// Stop recording the time it copy the images from global memory back to the host
	hipEventRecord(stop3);
	hipEventSynchronize(stop3);
	hipEventElapsedTime(&T3, start3, stop3);	

	// Start recording the time it takes to free up memory
	hipEventCreate(&start4);
	hipEventCreate(&stop4);
	hipEventRecord(start4);

	// Free up memory that is no longer needed
	hipFree(deviceGrayImage);
	hipFree(deviceRgbImage);
	hipFree(deviceHsvImage);

	// Stop recording the time it takes to free up memory
	hipEventRecord(stop4);
	hipEventSynchronize(stop4);
	hipEventElapsedTime(&T4, start4, stop4);

	// Stop recording the time for the entire function call
	hipEventRecord(stop0);
	hipEventSynchronize(stop0);
	hipEventElapsedTime(&T0, start0, stop0);

	// Write timing results
	printf("\n");
	printf("========= Gray Timing Details Start =========\n");
	printf("\n");
	printf(" \tTotal Function Time          : %f msec\n", T0);
	printf(" \tAllocate device memory time  : %f msec\n", T1);
	printf(" \tHost to device RGB copy time : %f msec\n", T1B);
	printf(" \tColor conversion kernel time : %f msec\n", T2);
	printf(" \tDevice to host copy time     : %f msec\n", T3);
	printf(" \tMemory free up time          : %f msec\n", T4);
	printf("\n");
	printf("========= Gray Timing Details End =========\n");
	printf("\n");
	hipProfilerStop();
}

void convertRGBToGrayAndHSV(const cv::Mat& input, cv::Mat& output1, cv::Mat& output2){
	int image_size = input.total();
	int width = input.cols;
	int height = input.rows;
	unsigned char* host_input = input.data;
	if (0 == output1.total()){
		output1.create(height, width, CV_8UC1);
	}
	if (0 == output2.total()){
		output2.create(height, width, CV_8UC3);
	}
	convertRGBToGrayAndHSV(host_input, output1.data, output2.data, width, height, 3);
}
